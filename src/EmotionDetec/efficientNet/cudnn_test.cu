#include <iostream>
#include <hipDNN.h>

#define CHECK_CUDNN(status) \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        std::cerr << "cuDNN error: " << hipdnnGetErrorString(status) << std::endl; \
        return 1; \
    }

int main() {
    hipdnnHandle_t cudnn;
    hipdnnStatus_t status = hipdnnCreate(&cudnn);
    CHECK_CUDNN(status);

    // Define tensor dimensions: (batch_size, channels, height, width)
    int batchSize = 1, channels = 1, height = 5, width = 5;
    int filterHeight = 3, filterWidth = 3;

    float input[batchSize][channels][height][width] = {{{{1, 2, 3, 4, 5},
                                                          {6, 7, 8, 9, 10},
                                                          {11, 12, 13, 14, 15},
                                                          {16, 17, 18, 19, 20},
                                                          {21, 22, 23, 24, 25}}}};

    float filter[channels][1][filterHeight][filterWidth] = {{{{1, 0, -1},
                                                              {1, 0, -1},
                                                              {1, 0, -1}}}};

    float output[batchSize][1][height - filterHeight + 1][width - filterWidth + 1];  // Output dimensions

    hipdnnTensorDescriptor_t input_desc, output_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;

    // Create tensor descriptors
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, channels, height, width));

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, 1, height - filterHeight + 1, width - filterWidth + 1));

    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filter_desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, channels, 1, filterHeight, filterWidth));

    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(conv_desc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    float alpha = 1.0f, beta = 0.0f;

    // Perform convolution
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha, input_desc, &input, filter_desc, &filter, conv_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, &beta, output_desc, &output));

    std::cout << "Convolution output: " << std::endl;
    for (int i = 0; i < height - filterHeight + 1; ++i) {
        for (int j = 0; j < width - filterWidth + 1; ++j) {
            std::cout << output[0][0][i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Cleanup
    hipdnnDestroy(cudnn);
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);

    return 0;
}
